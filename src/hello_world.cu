// hello_world.cu

#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel function
__global__ void helloWorld() {
    printf("Hello, World from GPU!\n");
}

int main() {
    // Launch the kernel with one block and one thread
    helloWorld<<<1, 1>>>();

    // Wait for the GPU to finish before accessing the results
    hipDeviceSynchronize();

    std::cout << "Hello, World from CPU!" << std::endl;

    return 0;
}
